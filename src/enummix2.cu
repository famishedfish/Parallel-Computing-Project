#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "enummix2.h"

struct GlobalConstants {
    cmatrix* A;
    cmatrix* B;
    int* q;
};

__constant__ GlobalConstants cuConstParams;
__constant__ int* theta;

void setup(cmatrix A, cmatrix B, int q) {
    GlobalConstants params;
    double *dataA, *dataB;

    hipMalloc(&dataA, sizeof(double) * A.getSize());
    hipMalloc(&dataB, sizeof(double) * A.getSize());
    hipMalloc(&params.A, sizeof(A));
    hipMalloc(&params.B, sizeof(B));
    hipMalloc(&params.q, sizeof(int));

    hipMemcpy(params.A, &A, sizeof(A), hipMemcpyHostToDevice);
    hipMemcpy(params.B, &B, sizeof(B), hipMemcpyHostToDevice);
    hipMemcpy(params.q, &q, sizeof(int), hipMemcpyHostToDevice);

    params.A->setx(dataA);
    params.A->setx(dataB);

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstParams), &params, sizeof(GlobalConstants));
}

int ENUMMIX(gnmgame &G, std::vector<std::vector<cvector> > &ans) {
    int n = G.getNumActions();
    cmatrix A = G.getPurePayoffMatrix(0);
    cmatrix B = G.getPurePayoffMatrix(1);

    unsigned int p1NumActions = G.getNumActions(0);
    unsigned int p2NumActions = G.getNumActions(1);
    unsigned int q = p1NumActions < p2NumActions ? p1NumActions : p2NumActions;

    setup(A, B, q);

    // theta = Generate(k, q)
    // theta = getKSupportSet(n, );
    // ans = Pure(A, B, q, theta)
    for (int k = 2; k <= q; k++) {
        // theta = Generate(k, q)
        // set up theta
        // run<<>>(k)
        // ans = ans union (A, B, k, q, theta)
    }

    // return 0;
}